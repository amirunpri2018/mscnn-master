#include "hip/hip_runtime.h"
namespace caffe {

  template <typename Dtype>
  __global__ void ROIPoolForward(const int nthreads, //线程数
                                 const Dtype* bottom_data,// 传入层数据指针
                                 const Dtype spatial_scale,//与原始图像相比,传入特征图的下采样倍数
                                 const int channels, //通道数
                                 const int height, //传入特征图高度
                                 const int width, //传入特征图宽度
                                 const int pooled_height, //传出特征图高度
                                 const int pooled_width, //传出特征图宽度
                                 const Dtype* bottom_rois, //传入层对应的ROI区域
                                 Dtype* top_data, //传出层数据指针
                                 int* argmax_data) //区域最大值位置索引
  {
    CUDA_KERNEL_LOOP(index, nthreads) {
      int pw = index % pooled_width; //表示传出数据对应的宽度索引
      int ph = (index / pooled_width) % pooled_height;//表示传出数据对应的高度索引
      int c = (index / pooled_width / pooled_height) % channels;//表示传出数据对应的通道索引
      int n = index / pooled_width / pooled_height / channels;//表示传出数据对应的batch索引
      //获取当前ROI在feature map上的位置
      bottom_rois += n * 5;
      int roi_batch_ind = bottom_rois[0];
      int roi_start_w = round(bottom_rois[1] * spatial_scale);
      int roi_start_h = round(bottom_rois[2] * spatial_scale);
      int roi_end_w = round(bottom_rois[3] * spatial_scale);
      int roi_end_h = round(bottom_rois[4] * spatial_scale);
      // 根据需要输出的feature map的大小,计算输入ROI中bin尺寸
      // 此时还是浮点数
      int roi_width = max(roi_end_w - roi_start_w + 1, 1);
      int roi_height = max(roi_end_h - roi_start_h + 1, 1);

      Dtype bin_size_h = static_cast<Dtype>(roi_height)
        / static_cast<Dtype>(pooled_height);
      Dtype bin_size_w = static_cast<Dtype>(roi_width)
        / static_cast<Dtype>(pooled_width);
      //获取bin的起始结束坐标,并做保护措施,注意起始坐标获取采用floor函数
      //即向下取整,结束坐标获取采用ceil函数,即向上取整,即会使得相邻两个bin有
      //重叠
      int hstart = static_cast<int>(floor(static_cast<Dtype>(ph)
        * bin_size_h));
      int wstart = static_cast<int>(floor(static_cast<Dtype>(pw)
        * bin_size_w));
      int hend = static_cast<int>(ceil(static_cast<Dtype>(ph + 1)
        * bin_size_h));
      int wend = static_cast<int>(ceil(static_cast<Dtype>(pw + 1)
        * bin_size_w));
      // (Add roi offsets and clip to input boundaries)
      hstart = min(max(hstart + roi_start_h, 0), height);
      hend = min(max(hend + roi_start_h, 0), height);
      wstart = min(max(wstart + roi_start_w, 0), width);
      wend = min(max(wend + roi_start_w, 0), width);
      bool is_empty = (hend <= hstart) || (wend <= wstart);

      // maxval用于记录每一个小块的最大值
      Dtype maxval = is_empty ? 0 : -FLT_MAX;
      // (If nothing is pooled, argmax = -1 causes nothing to be backprop'd)
      // maxidx 用于记录当前bin的最大值索引位置,默认值-1
      int maxidx = -1;
      // 根据传入参数将bottom_data偏移到感兴趣的那一层
      bottom_data += (roi_batch_ind * channels + c) * height * width;
      // 操作感兴趣层对应bin的数据,获取最大值,并记录最大值位置
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          int bottom_index = h * width + w;
          if (bottom_data[bottom_index] > maxval) {
            maxval = bottom_data[bottom_index];
            maxidx = bottom_index;
          }
        }
      }
      top_data[index] = maxval;
      argmax_data[index] = maxidx;
    }
  }

  template <typename Dtype>
  void ROIPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* bottom_rois = bottom[1]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    int* argmax_data = max_idx_.mutable_gpu_data();
    int count = top[0]->count();//前向运算的count数是由传出feature map的尺寸决定
    // NOLINT_NEXT_LINE(whitespace/operators)
    ROIPoolForward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
      count, bottom_data, spatial_scale_, channels_, height_, width_,
      pooled_height_, pooled_width_, bottom_rois, top_data, argmax_data);
    CUDA_POST_KERNEL_CHECK;
  }

  template <typename Dtype>
  __global__ void ROIPoolBackward(const int nthreads, //线程数
                                  const Dtype* top_diff, //传出层特征图梯度值
                                  const int* argmax_data, //区域最大值位置索引
                                  const int num_rois, //roi的个数
                                  const Dtype spatial_scale, //与原始图像相比,传入特征图的下采样倍数
                                  const int channels, //传入层通道数
                                  const int height, //传入层特征图高度
                                  const int width, //传入层特征图宽度
                                  const int pooled_height, //传出特征图高度
                                  const int pooled_width, //传出特征图宽度
                                  Dtype* bottom_diff, //传入层特征图梯度值
                                  const Dtype* bottom_rois) {//传入层ROI数据
    CUDA_KERNEL_LOOP(index, nthreads) {
      // (n, c, h, w) coords in bottom data
      int w = index % width;//表示传入数据对应的宽度索引
      int h = (index / width) % height;//表示传入数据对应的高度索引
      int c = (index / width / height) % channels;//表示传入数据对应的通道数索引
      int n = index / width / height / channels;//表示传入数据对应的batch索引

      Dtype gradient = 0;
      // Accumulate gradient over all ROIs that pooled this element
      // 累加所有ROI中跟这一点有关的梯度值
      for (int roi_n = 0; roi_n < num_rois; ++roi_n) {
        const Dtype* offset_bottom_rois = bottom_rois + roi_n * 5;
        int roi_batch_ind = offset_bottom_rois[0];
        // Skip if ROI's batch index doesn't match n
        if (n != roi_batch_ind) {
          continue;
        }

        int roi_start_w = round(offset_bottom_rois[1] * spatial_scale);
        int roi_start_h = round(offset_bottom_rois[2] * spatial_scale);
        int roi_end_w = round(offset_bottom_rois[3] * spatial_scale);
        int roi_end_h = round(offset_bottom_rois[4] * spatial_scale);

        // Skip if ROI doesn't include (h, w)
        //如果此点不在当前ROI内,则继续循环,若所有ROI都不包含此点,则此点梯度值为0
        const bool in_roi = (w >= roi_start_w && w <= roi_end_w &&
          h >= roi_start_h && h <= roi_end_h);
        if (!in_roi) {
          continue;
        }

        int offset = (roi_n * channels + c) * pooled_height * pooled_width;
        const Dtype* offset_top_diff = top_diff + offset;
        const int* offset_argmax_data = argmax_data + offset;

        // Compute feasible set of pooled units that could have pooled
        // this bottom unit

        // Force malformed ROIs to be 1x1

        // 以下操作是根据当前传入层数据的坐标查找传出层中对应位置的数据,并记录填充
        // 若当前传入层数据点对一个以上ROI做了贡献,则梯度累加
        int roi_width = max(roi_end_w - roi_start_w + 1, 1);
        int roi_height = max(roi_end_h - roi_start_h + 1, 1);
        Dtype bin_size_h = static_cast<Dtype>(roi_height)
          / static_cast<Dtype>(pooled_height);
        Dtype bin_size_w = static_cast<Dtype>(roi_width)
          / static_cast<Dtype>(pooled_width);

        int phstart = floor(static_cast<Dtype>(h - roi_start_h) / bin_size_h);
        int phend = ceil(static_cast<Dtype>(h - roi_start_h + 1) / bin_size_h);
        int pwstart = floor(static_cast<Dtype>(w - roi_start_w) / bin_size_w);
        int pwend = ceil(static_cast<Dtype>(w - roi_start_w + 1) / bin_size_w);

        phstart = min(max(phstart, 0), pooled_height);
        phend = min(max(phend, 0), pooled_height);
        pwstart = min(max(pwstart, 0), pooled_width);
        pwend = min(max(pwend, 0), pooled_width);

        for (int ph = phstart; ph < phend; ++ph) {
          for (int pw = pwstart; pw < pwend; ++pw) {
            if (offset_argmax_data[ph * pooled_width + pw] == (h * width + w)) {
              gradient += offset_top_diff[ph * pooled_width + pw];
            }
          }
        }
      }
      bottom_diff[index] = gradient;
    }
  }

  template <typename Dtype>
  void ROIPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    if (!propagate_down[0]) {
      return;
    }
    // bottom和top的相对关系由forward决定
    const Dtype* bottom_rois = bottom[1]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();//前向运算的count数是由传入feature map的尺寸决定
    caffe_gpu_set(count, Dtype(0.), bottom_diff);
    const int* argmax_data = max_idx_.gpu_data();
    // NOLINT_NEXT_LINE(whitespace/operators)
    ROIPoolBackward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
      count, top_diff, argmax_data, top[0]->num(), spatial_scale_, channels_,
      height_, width_, pooled_height_, pooled_width_, bottom_diff, bottom_rois);
    CUDA_POST_KERNEL_CHECK;
  }

  INSTANTIATE_LAYER_GPU_FUNCS(ROIPoolingLayer);

}  // namespace caffe
